#include "hip/hip_runtime.h"
/*
 * This file is part of Kintinuous.
 *
 * Copyright (C) 2015 The National University of Ireland Maynooth and 
 * Massachusetts Institute of Technology
 *
 * The use of the code within this file and all code within files that 
 * make up the software that is Kintinuous is permitted for 
 * non-commercial purposes only.  The full terms and conditions that 
 * apply to the code within this file are detailed within the LICENSE.txt 
 * file and at <http://www.cs.nuim.ie/research/vision/data/kintinuous/code.php> 
 * unless explicitly stated.  By downloading this file you agree to 
 * comply with these terms.
 *
 * If you wish to use any of this code for commercial purposes then 
 * please email commercialisation@nuim.ie.
 *
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 *
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"

__device__ __forceinline__ float
getMinTime (const float3& volume_max, const float3& origin, const float3& dir)
{
  float txmin = ( (dir.x > 0 ? 0.f : volume_max.x) - origin.x) / dir.x;
  float tymin = ( (dir.y > 0 ? 0.f : volume_max.y) - origin.y) / dir.y;
  float tzmin = ( (dir.z > 0 ? 0.f : volume_max.z) - origin.z) / dir.z;

  return fmax ( fmax (txmin, tymin), tzmin);
}

__device__ __forceinline__ float
getMaxTime (const float3& volume_max, const float3& origin, const float3& dir)
{
  float txmax = ( (dir.x > 0 ? volume_max.x : 0.f) - origin.x) / dir.x;
  float tymax = ( (dir.y > 0 ? volume_max.y : 0.f) - origin.y) / dir.y;
  float tzmax = ( (dir.z > 0 ? volume_max.z : 0.f) - origin.z) / dir.z;

  return fmin (fmin (txmax, tymax), tzmax);
}

struct RayCaster
{
  enum { CTA_SIZE_X = 32, CTA_SIZE_Y = 8 };

  Mat33 Rcurr;
  float3 tcurr;

  float time_step;
  float3 volume_size;

  float3 cell_size;
  int cols, rows;

  PtrStep<short> volume;

  Intr intr;

  mutable PtrStep<float> nmap;
  mutable PtrStep<float> vmap;

  int3 voxelWrap;

  mutable PtrStep<uchar4> vmap_curr_color;
  PtrStep<uchar4> color_volume;

  __device__ __forceinline__ float3
  get_ray_next (int x, int y) const
  {
    float3 ray_next;
    ray_next.x = (x - intr.cx) / intr.fx;
    ray_next.y = (y - intr.cy) / intr.fy;
    ray_next.z = 1;
    return ray_next;
  }

  __device__ __forceinline__ bool
  checkInds (const int3& g) const
  {
    return (g.x >= 0 && g.y >= 0 && g.z >= 0 && g.x < VOLUME_X && g.y < VOLUME_Y && g.z < VOLUME_X);
  }

  __device__ __forceinline__ float
  readTsdf (int x, int y, int z) const
  {
      const short * pos = &volume.ptr(0)[((x + voxelWrap.x) % VOLUME_X) + ((y + voxelWrap.y) % VOLUME_Y) * VOLUME_X + ((z + voxelWrap.z) % VOLUME_Z) * VOLUME_X * VOLUME_Y];
      return unpack_tsdf (*pos);
  }

  __device__ __forceinline__ float
  readHeat (int x, int y, int z) const
  {
      const uchar4 * ptrColor = &color_volume.ptr(0)[((x + voxelWrap.x) % VOLUME_X) + ((y + voxelWrap.y) % VOLUME_Y) * VOLUME_X + ((z + voxelWrap.z) % VOLUME_Z) * VOLUME_X * VOLUME_Y];
      return ptrColor->w;
  }

  __device__ __forceinline__ float
  readRed (int x, int y, int z) const
  {
      const uchar4 * ptrColor = &color_volume.ptr(0)[((x + voxelWrap.x) % VOLUME_X) + ((y + voxelWrap.y) % VOLUME_Y) * VOLUME_X + ((z + voxelWrap.z) % VOLUME_Z) * VOLUME_X * VOLUME_Y];
      return ptrColor->x;
  }

  __device__ __forceinline__ float
  readGreen (int x, int y, int z) const
  {
      const uchar4 * ptrColor = &color_volume.ptr(0)[((x + voxelWrap.x) % VOLUME_X) + ((y + voxelWrap.y) % VOLUME_Y) * VOLUME_X + ((z + voxelWrap.z) % VOLUME_Z) * VOLUME_X * VOLUME_Y];
      return ptrColor->y;
  }

  __device__ __forceinline__ float
  readBlue (int x, int y, int z) const
  {
      const uchar4 * ptrColor = &color_volume.ptr(0)[((x + voxelWrap.x) % VOLUME_X) + ((y + voxelWrap.y) % VOLUME_Y) * VOLUME_X + ((z + voxelWrap.z) % VOLUME_Z) * VOLUME_X * VOLUME_Y];
      return ptrColor->z;
  }

  __device__ __forceinline__ int3
  getVoxel (float3 point) const
  {
    int vx = __float2int_rd (point.x / cell_size.x);        // round to negative infinity
    int vy = __float2int_rd (point.y / cell_size.y);
    int vz = __float2int_rd (point.z / cell_size.z);

    return make_int3 (vx, vy, vz);
  }

  __device__ __forceinline__ float
  interpolateTrilineary (const float3& origin, const float3& dir, float time) const
  {
    return interpolateTrilineary (origin + dir * time);
  }

  __device__ __forceinline__ float
  interpolateTrilineary (const float3& point) const
  {
    int3 g = getVoxel (point);

    if (g.x <= 0 || g.x >= VOLUME_X - 1)
      return numeric_limits<float>::quiet_NaN ();

    if (g.y <= 0 || g.y >= VOLUME_Y - 1)
      return numeric_limits<float>::quiet_NaN ();

    if (g.z <= 0 || g.z >= VOLUME_Z - 1)
      return numeric_limits<float>::quiet_NaN ();

    float vx = (g.x + 0.5f) * cell_size.x;
    float vy = (g.y + 0.5f) * cell_size.y;
    float vz = (g.z + 0.5f) * cell_size.z;

    g.x = (point.x < vx) ? (g.x - 1) : g.x;
    g.y = (point.y < vy) ? (g.y - 1) : g.y;
    g.z = (point.z < vz) ? (g.z - 1) : g.z;

    float a = (point.x - (g.x + 0.5f) * cell_size.x) / cell_size.x;
    float b = (point.y - (g.y + 0.5f) * cell_size.y) / cell_size.y;
    float c = (point.z - (g.z + 0.5f) * cell_size.z) / cell_size.z;

    float res = readTsdf (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
                readTsdf (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
                readTsdf (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
                readTsdf (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
                readTsdf (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
                readTsdf (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
                readTsdf (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
                readTsdf (g.x + 1, g.y + 1, g.z + 1) * a * b * c;
    return res;
  }

  __device__ __forceinline__ uchar3
  interpolateColorTrilineary (const float3& point) const
  {
    int3 g = getVoxel (point);

    uchar3 black = {0, 0, 0};

    if (g.x <= 0 || g.x >= VOLUME_X - 1)
      return black;

    if (g.y <= 0 || g.y >= VOLUME_Y - 1)
      return black;

    if (g.z <= 0 || g.z >= VOLUME_Z - 1)
      return black;

    float vx = (g.x + 0.5f) * cell_size.x;
    float vy = (g.y + 0.5f) * cell_size.y;
    float vz = (g.z + 0.5f) * cell_size.z;

    g.x = (point.x < vx) ? (g.x - 1) : g.x;
    g.y = (point.y < vy) ? (g.y - 1) : g.y;
    g.z = (point.z < vz) ? (g.z - 1) : g.z;

    float a = (point.x - (g.x + 0.5f) * cell_size.x) / cell_size.x;
    float b = (point.y - (g.y + 0.5f) * cell_size.y) / cell_size.y;
    float c = (point.z - (g.z + 0.5f) * cell_size.z) / cell_size.z;

    uchar3 res = {readRed (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
                  readRed (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
                  readRed (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
                  readRed (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
                  readRed (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
                  readRed (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
                  readRed (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
                  readRed (g.x + 1, g.y + 1, g.z + 1) * a * b * c,
                  readGreen (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
                  readGreen (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
                  readGreen (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
                  readGreen (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
                  readGreen (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
                  readGreen (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
                  readGreen (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
                  readGreen (g.x + 1, g.y + 1, g.z + 1) * a * b * c,
                  readBlue (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
                  readBlue (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
                  readBlue (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
                  readBlue (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
                  readBlue (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
                  readBlue (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
                  readBlue (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
                  readBlue (g.x + 1, g.y + 1, g.z + 1) * a * b * c};

    return res;
  }

  __device__ __forceinline__ float
  interpolateHeatTrilineary (const float3& point) const
  {
    int3 g = getVoxel (point);

    if (g.x <= 0 || g.x >= VOLUME_X - 1)
      return numeric_limits<float>::quiet_NaN ();

    if (g.y <= 0 || g.y >= VOLUME_Y - 1)
      return numeric_limits<float>::quiet_NaN ();

    if (g.z <= 0 || g.z >= VOLUME_Z - 1)
      return numeric_limits<float>::quiet_NaN ();

    float vx = (g.x + 0.5f) * cell_size.x;
    float vy = (g.y + 0.5f) * cell_size.y;
    float vz = (g.z + 0.5f) * cell_size.z;

    g.x = (point.x < vx) ? (g.x - 1) : g.x;
    g.y = (point.y < vy) ? (g.y - 1) : g.y;
    g.z = (point.z < vz) ? (g.z - 1) : g.z;

    float a = (point.x - (g.x + 0.5f) * cell_size.x) / cell_size.x;
    float b = (point.y - (g.y + 0.5f) * cell_size.y) / cell_size.y;
    float c = (point.z - (g.z + 0.5f) * cell_size.z) / cell_size.z;

    float res = readHeat (g.x + 0, g.y + 0, g.z + 0) * (1 - a) * (1 - b) * (1 - c) +
                readHeat (g.x + 0, g.y + 0, g.z + 1) * (1 - a) * (1 - b) * c +
                readHeat (g.x + 0, g.y + 1, g.z + 0) * (1 - a) * b * (1 - c) +
                readHeat (g.x + 0, g.y + 1, g.z + 1) * (1 - a) * b * c +
                readHeat (g.x + 1, g.y + 0, g.z + 0) * a * (1 - b) * (1 - c) +
                readHeat (g.x + 1, g.y + 0, g.z + 1) * a * (1 - b) * c +
                readHeat (g.x + 1, g.y + 1, g.z + 0) * a * b * (1 - c) +
                readHeat (g.x + 1, g.y + 1, g.z + 1) * a * b * c;
    return res;
  }

  __device__ __forceinline__ void
  operator () () const
  {
    int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
    int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

    if (x >= cols || y >= rows)
      return;

    vmap.ptr (y)[x] = numeric_limits<float>::quiet_NaN ();
    nmap.ptr (y)[x] = numeric_limits<float>::quiet_NaN ();

    float3 ray_start = tcurr;
    float3 ray_next = Rcurr * get_ray_next (x, y) + tcurr;

    float3 ray_dir = normalized (ray_next - ray_start);

    //ensure that it isn't a degenerate case
    ray_dir.x = (ray_dir.x == 0.f) ? 1e-15 : ray_dir.x;
    ray_dir.y = (ray_dir.y == 0.f) ? 1e-15 : ray_dir.y;
    ray_dir.z = (ray_dir.z == 0.f) ? 1e-15 : ray_dir.z;

    // computer time when entry and exit volume
    float time_start_volume = getMinTime (volume_size, ray_start, ray_dir);
    float time_exit_volume = getMaxTime (volume_size, ray_start, ray_dir);

    const float min_dist = 0.f;         //in meters
    time_start_volume = fmax (time_start_volume, min_dist);
    if (time_start_volume >= time_exit_volume)
      return;

    float time_curr = time_start_volume;
    int3 g = getVoxel (ray_start + ray_dir * time_curr);
    g.x = max (0, min (g.x, VOLUME_X - 1));
    g.y = max (0, min (g.y, VOLUME_Y - 1));
    g.z = max (0, min (g.z, VOLUME_Z - 1));

    float tsdf = readTsdf (g.x, g.y, g.z);

    //infinite loop guard
    const float max_time = 3 * (volume_size.x + volume_size.y + volume_size.z);

    for (; time_curr < max_time; time_curr += time_step)
    {
      float tsdf_prev = tsdf;

      int3 g = getVoxel (  ray_start + ray_dir * (time_curr + time_step)  );
      if (!checkInds (g))
        break;

      tsdf = readTsdf (g.x, g.y, g.z);

      if (tsdf_prev < 0.f && tsdf > 0.f)
        break;

      if (tsdf_prev > 0.f && tsdf < 0.f)           //zero crossing
      {
        float Ftdt = interpolateTrilineary (ray_start, ray_dir, time_curr + time_step);
        if (isnan (Ftdt))
          break;

        float Ft = interpolateTrilineary (ray_start, ray_dir, time_curr);
        if (isnan (Ft))
          break;

        float Ts = time_curr - time_step * Ft / (Ftdt - Ft);

        float3 vetex_found = ray_start + ray_dir * Ts;

        vmap.ptr (y       )[x] = vetex_found.x;
        vmap.ptr (y + rows)[x] = vetex_found.y;
        vmap.ptr (y + 2 * rows)[x] = vetex_found.z;

        int3 g = getVoxel ( ray_start + ray_dir * time_curr );

        uchar3 pointColor = interpolateColorTrilineary(vetex_found);

        vmap_curr_color.ptr(y)[x].x = pointColor.x;
        vmap_curr_color.ptr(y)[x].y = pointColor.y;
        vmap_curr_color.ptr(y)[x].z = pointColor.z;
        vmap_curr_color.ptr(y)[x].w = interpolateHeatTrilineary(vetex_found);

        if (g.x > 1 && g.y > 1 && g.z > 1 && g.x < VOLUME_X - 2 && g.y < VOLUME_Y - 2 && g.z < VOLUME_Z - 2)
        {
          float3 t;
          float3 n;

          t = vetex_found;
          t.x += cell_size.x;
          float Fx1 = interpolateTrilineary (t);

          t = vetex_found;
          t.x -= cell_size.x;
          float Fx2 = interpolateTrilineary (t);

          n.x = (Fx1 - Fx2);

          t = vetex_found;
          t.y += cell_size.y;
          float Fy1 = interpolateTrilineary (t);

          t = vetex_found;
          t.y -= cell_size.y;
          float Fy2 = interpolateTrilineary (t);

          n.y = (Fy1 - Fy2);

          t = vetex_found;
          t.z += cell_size.z;
          float Fz1 = interpolateTrilineary (t);

          t = vetex_found;
          t.z -= cell_size.z;
          float Fz2 = interpolateTrilineary (t);

          n.z = (Fz1 - Fz2);

          n = normalized (n);

          nmap.ptr (y       )[x] = n.x;
          nmap.ptr (y + rows)[x] = n.y;
          nmap.ptr (y + 2 * rows)[x] = n.z;
        }
        break;
      }

    }          /* for(;;)  */
  }
};

__global__ void
rayCastKernel (const RayCaster rc) {
  rc ();
}

void
raycast (const Intr& intr, const Mat33& Rcurr, const float3& tcurr, 
                      float tranc_dist, const float3& volume_size,
                      const PtrStep<short>& volume, DeviceArray2D<float>& vmap, DeviceArray2D<float>& nmap, const int3 & voxelWrap, DeviceArray2D<uchar4> & vmap_curr_color,
                      PtrStep<uchar4> color_volume)
{
  RayCaster rc;

  rc.Rcurr = Rcurr;
  rc.tcurr = tcurr;

  rc.time_step = tranc_dist * 0.8f;

  rc.volume_size = volume_size;

  rc.cell_size.x = volume_size.x / VOLUME_X;
  rc.cell_size.y = volume_size.y / VOLUME_Y;
  rc.cell_size.z = volume_size.z / VOLUME_Z;

  rc.cols = vmap.cols ();
  rc.rows = vmap.rows () / 3;

  rc.intr = intr;

  rc.volume = volume;
  rc.vmap = vmap;
  rc.nmap = nmap;

  rc.voxelWrap = voxelWrap;

  rc.vmap_curr_color = vmap_curr_color;
  rc.color_volume = color_volume;

  dim3 block (RayCaster::CTA_SIZE_X, RayCaster::CTA_SIZE_Y);
  dim3 grid (divUp (rc.cols, block.x), divUp (rc.rows, block.y));

  rayCastKernel<<<grid, block>>>(rc);
  cudaSafeCall (hipGetLastError ());
}

